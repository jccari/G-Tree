#include "cudaFun.h"
#include <cuda/hip/hip_runtime.h>
#include <malloc.h>

void allocateArray(float* h_arr, float* &d_arr, long totalsize){
    size_t sizecpy = totalsize * sizeof(float);
    hipMalloc((void **) &d_arr, sizecpy);
    hipMemcpy(d_arr, h_arr, sizecpy, hipMemcpyHostToDevice);
}

void deallocateArray(float* &h_arr, float* d_arr, long totalsize){
    size_t sizecpy = totalsize * sizeof(float);
    h_arr = (float *)malloc(totalsize * sizeof(float));
    hipMemcpy(h_arr, d_arr, sizecpy, hipMemcpyDeviceToHost);
}

void deleteDeviceArray(float *&d_arr, long totalsize){
    hipFree(d_arr);
}
#include "hip/hip_runtime.h"
#include <stdio.h>
#define GPU_THREADS 	512 // check this
#define NUM_DIMS 		10
#define Intersect_Flags int
#define float 	float*

int fanout = 2;

//float -> is array of children nodes

__device__ // __global__ ?
bool check_intersection_leaf(int pos_node, float* query, float* nodes_tree, int innodes, totalnodes){
	bool intersect = false;
	int ini_children = ini(i, totalnodes, 0, fanout, NUM_DIMS, innodes, totalnodes);
	for (int i = 0; i < NUM_DIMS; ++i){
		if ( nodes_tree[ini_children + i] == query[i] )
			intersect = true
	}

	return intersect;
}

__device__ // __global__ ?
bool check_intersection_internal(int pos_node, float* query, float* nodes_tree, int innodes, totalnodes){
	bool intersect = false;
	int ini_children = ini(i, totalnodes, 0, fanout, NUM_DIMS, innodes, totalnodes);
	for (int i = 0; i < 2; ++i){
		if ( nodes_tree[ini_children] < query[0] && nodes_tree[ini_children+1] > query[1] )
			intersect = true
	}

	return intersect;
}


__device__
//void makeIntersections(float* query, float* offset, Intersect_Flags* intersect_flags, int i, int totalnodes){
void makeIntersections(float* query, long pos_offset, int &intersect_flag, int i, int numChildren, float* nodes_tree, bool isInternal, int innodes, int totalnodes){
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	//int row = threadIdx.y + blockIdx.y*blockDim.y;

	if (isInternal)
		if ( idx < numChildren && check_intersection_internal( pos_offset, query, nodes_tree, innodes, totalnodes))
			intersect_flag = 1;
		else intersect_flag = 0;
	else
		if ( idx < numChildren && check_intersection_leaf( pos_offset, query, nodes_tree, innodes, totalnodes))
			intersect_flag = 1;
		else intersect_flag = 0;

}

// definir 
	// fanout
	// dim -> NUM_DIMS
	// innodes

__global__
void intersectionKernel( float* query, float* offsets, int*& intersect_flags, int totalnodes, float* nodes_tree, int innodes){
	dim3 dimBlock( GPU_THREADS, 1, 1);
	dim3 dimGrid( ceil(numEntries/GPU_THREADS) , 1, 1);

	long* pos_children;
	children(i, pos_children, fanout, totalnodes);
	int num_children = (int) sizeof(pos_children)/ sizeof(long);
	//float* children = children();
	int ini_children = ini(i, totalnodes, 0, fanout, NUM_DIMS, innodes, totalnodes);
	int fin_children = fin(i, totalnodes, 0, fanout, NUM_DIMS, innodes, totalnodes);

	for (int i = 0; i < num_children; ++i){
		if ( isInternalNode(i) )
			makeIntersections<<< dimGrid, dimBlock>>> (query, pos_children[i], intersect_flags[i], i, num_children, nodes_tree, 1, innodes, totalnodes);
		else
			makeIntersections<<< dimGrid, dimBlock>>> (query, pos_children[i], intersect_flags[i], i, num_children, nodes_tree, 0, innodes, totalnodes);
			//makeIntersections<<< dimGrid, dimBlock>>> (query, offsets[i], intersect_flags[i], i, totalnodes);
	}
	
	//TODO: copy offsets[i]' childen to offsets
	for (int i = 0; i < num_children; ++i){
		offsets[i] = nodes_tree[ ini_children+i ];
	}
}


/*Eliminate entries that have not intersection with Q query*/
__host__
void compactKernel(float* offsets, int*& intersect_flags, int size){
	for (int i = 0; i < size; ++i){
		if( intersect_flags[i] == 0)
			offsets[i] = 0;

		intersect_flags[i] = 0;
	}
}

__global__
float* rangeQuery(float* root, float* query, float* nodes_tree ,int totalnodes, int innodes, int height){
	int* offsets = (int*) malloc( totalnodes*sizeof(int) );
	offsets[0] = 0; // inicializar con root que esta en la posicion 0

	int* d_offsets = (int*) malloc( sizeof(int) );
	//int numNodes = (int) sizeof(offsets)/ sizeof(float);
	int* intersect_flags = (int*) malloc( totalnodes*sizeof(int) ); // inicializar en -1

	hipMalloc( (void**)& d_offsets, sizeof(int));
 	hipMemcpy( d_offsets, &offsets, sizeof(int), hipMemcpyHostToDevice) 

 	int size;
	for (int i = 0; i < height; ++i){
		intersectionKernel( query, offsets, intersect_flags, totalnodes, nodes_tree, innodes);
		size = (int) sizeof(intersect_flags)/sizeof(int);
		compactKernel( offsets, intersect_flags, size);
	}
}